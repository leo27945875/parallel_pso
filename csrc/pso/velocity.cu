#include "hip/hip_runtime.h"
#include "velocity.cuh"
#include "utils.cuh"


__global__ void update_velocities_kernel(
    scalar_t       *vs, 
    scalar_t const *xs, 
    scalar_t const *local_best_xs, 
    scalar_t const *global_best_x,
    scalar_t        w,
    scalar_t        c0,
    scalar_t        c1,
    ssize_t         num, 
    ssize_t         dim,
    cuda_rng_t     *rng_states
){
    ssize_t nid = blockIdx.x * blockDim.x + threadIdx.x;
    ssize_t idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (nid >= num)
        return;
        
    for (ssize_t i = idx; i < dim; i += gridDim.y * blockDim.y){
        // Load data into local memory:
        scalar_t v = vs[nid * dim + i];
        scalar_t x = xs[nid * dim + i];
        scalar_t lbest_x = local_best_xs[nid * dim + i];
        scalar_t gbest_x = global_best_x[i];

        // Update velocities:
#if IS_VELOVITY_USE_RANDOM
        cuda_rng_t thread_rng_state = rng_states[nid * dim + i];
        vs[nid * dim + i] = (
            w * v + 
            c0 * _get_curand_uniform<scalar_t>(&thread_rng_state) * (lbest_x - x) + 
            c1 * _get_curand_uniform<scalar_t>(&thread_rng_state) * (gbest_x - x)
        );
        rng_states[nid * dim + idx] = thread_rng_state;
#else
        vs[nid * dim + i] = (
            w * v + 
            c0 * (lbest_x - x) + 
            c1 * (gbest_x - x)
        );
#endif
    }
}

__global__ void update_velocities_with_sum_pow2_kernel(
    scalar_t       *vs, 
    scalar_t const *xs, 
    scalar_t const *local_best_xs, 
    scalar_t const *global_best_x,
    scalar_t       *v_sum_pow2_res,
    scalar_t        w,
    scalar_t        c0,
    scalar_t        c1,
    ssize_t         num, 
    ssize_t         dim,
    cuda_rng_t     *rng_states
){
    __shared__ scalar_t p_smem[BLOCK_DIM_X][BLOCK_DIM_Y];

    ssize_t nid = blockIdx.x * blockDim.x + threadIdx.x;
    ssize_t idx = blockIdx.y * blockDim.y + threadIdx.y;
    ssize_t tidx = threadIdx.x;
    ssize_t tidy = threadIdx.y;

    p_smem[tidx][tidy] = 0.;

    if (nid >= num)
        return;

    for (ssize_t i = idx; i < dim; i += gridDim.y * blockDim.y){
        // Load data into local memory:
        scalar_t v = vs[nid * dim + i];
        scalar_t x = xs[nid * dim + i];
        scalar_t lbest_x = local_best_xs[nid * dim + i];
        scalar_t gbest_x = global_best_x[i];

        // Calculate new velocities:
#if IS_VELOVITY_USE_RANDOM
        cuda_rng_t thread_rng_state = rng_states[nid * dim + idx];
        v = (
            w * v + 
            c0 * _get_curand_uniform<scalar_t>(&thread_rng_state) * (lbest_x - x) + 
            c1 * _get_curand_uniform<scalar_t>(&thread_rng_state) * (gbest_x - x)
        );
        rng_states[nid * dim + idx] = thread_rng_state;
#else
        v = (
            w * v + 
            c0 * (lbest_x - x) + 
            c1 * (gbest_x - x)
        );
#endif
        // Store v^2 into shared memory:
        p_smem[tidx][tidy] += v * v;

        // Store v into global memory:
        vs[nid * dim + i] = v;
    }
    // Sum the squares:
    for (ssize_t k = blockDim.y >> 1; k > 0; k >>= 1){
        __syncthreads();
        if (tidy < k)
            p_smem[tidx][tidy] += p_smem[tidx][tidy + k];
    }
    if (tidy == 0)
        atomicAdd(v_sum_pow2_res + nid, p_smem[tidx][0]);
}

__global__ void norm_clip_velocities_kernel(
    scalar_t *vs, 
    scalar_t *v_sum_pow2_res,
    scalar_t  v_max,
    ssize_t   num, 
    ssize_t   dim
){
    ssize_t nid = blockIdx.x * blockDim.x + threadIdx.x;
    ssize_t idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (nid >= num)
        return;

    scalar_t norm = sqrt(v_sum_pow2_res[nid]); // 'Broadcast' mechanism (see https://forums.developer.nvidia.com/t/accessing-same-global-memory-address-within-warps/66574)
    if (norm <= v_max)
        return;

    scalar_t scale = v_max / norm;
    for (ssize_t i = idx; i < dim; i += gridDim.y * blockDim.y){
        vs[nid * dim + i] *= scale;
    }
}

void update_velocities_cuda(
    scalar_t       *vs_cuda_ptr, 
    scalar_t const *xs_cuda_ptr, 
    scalar_t const *local_best_xs_cuda_ptr, 
    scalar_t const *global_best_x_cuda_ptr,
    scalar_t       *v_sum_pow2_cuda_ptr,
    scalar_t        w,
    scalar_t        c0,
    scalar_t        c1,
    scalar_t        v_max,
    ssize_t         num, 
    ssize_t         dim,
    cuda_rng_t     *rng_states_cuda_ptr
){
    dim3 grid_dims(get_num_block_x(num), get_num_block_y(dim));
    dim3 block_dims(BLOCK_DIM_X, BLOCK_DIM_Y);

    if (v_max <= 0.){
        update_velocities_kernel<<<grid_dims, block_dims>>>(
            vs_cuda_ptr, xs_cuda_ptr, local_best_xs_cuda_ptr, global_best_x_cuda_ptr, w, c0, c1, num, dim, rng_states_cuda_ptr
        );
        cudaCheckErrors("Failed to run 'update_velocities_kernel'.");
        
    }else{
        bool is_v_sum_pow2_no_buffer = (v_sum_pow2_cuda_ptr == nullptr);
        if (is_v_sum_pow2_no_buffer){
            hipMalloc(&v_sum_pow2_cuda_ptr, num * sizeof(scalar_t));
            cudaCheckErrors("Failed to allocate memory buffer to 'v_sum_pow2_cuda_ptr'.");
        }
        hipMemset(v_sum_pow2_cuda_ptr, 0, num * sizeof(scalar_t));
        cudaCheckErrors("Failed to set buffer 'v_sum_pow2_cuda_ptr' to zeros.");

        update_velocities_with_sum_pow2_kernel<<<grid_dims, block_dims>>>(
            vs_cuda_ptr, xs_cuda_ptr, local_best_xs_cuda_ptr, global_best_x_cuda_ptr, v_sum_pow2_cuda_ptr, w, c0, c1, num, dim, rng_states_cuda_ptr
        );
        cudaCheckErrors("Failed to run 'update_velocities_with_sum_pow2_kernel'.");
        
        norm_clip_velocities_kernel<<<grid_dims, block_dims>>>(
            vs_cuda_ptr, v_sum_pow2_cuda_ptr, v_max, num, dim
        );
        cudaCheckErrors("Failed to run 'norm_clip_velocities_kernel'.");

        if (is_v_sum_pow2_no_buffer){
            hipFree(v_sum_pow2_cuda_ptr);
            cudaCheckErrors("Failed to free memory buffer to 'v_sum_pow2_cuda_ptr'.");
        }
    }
}