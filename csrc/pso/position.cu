#include "hip/hip_runtime.h"
#include "position.cuh"
#include "utils.cuh"


__global__ void update_positions_kernel(
    scalar_t       *xs,
    scalar_t const *vs,
    scalar_t        x_min,
    scalar_t        x_max,
    ssize_t         num,
    ssize_t         dim
){
    ssize_t nid = blockIdx.x;
    ssize_t idx = blockIdx.y * blockDim.x + threadIdx.x;
    for (ssize_t i = idx; i < dim; i += gridDim.y * blockDim.x)
        xs[nid * dim + i] = min(
            x_max, 
            max(
                x_min, 
                xs[nid * dim + i] + vs[nid * dim + i]
            )
        );
}

void update_positions_cuda(
    scalar_t       *xs_cuda_ptr,
    scalar_t const *vs_cuda_ptr,
    scalar_t        x_min,
    scalar_t        x_max,
    ssize_t         num,
    ssize_t         dim
){
    dim3 grid_dims(num, get_num_block_1d(dim));
    dim3 block_dims(BLOCK_DIM_1D);
    update_positions_kernel<<<grid_dims, block_dims>>>(
        xs_cuda_ptr, vs_cuda_ptr, x_min, x_max, num, dim
    );
    cudaCheckErrors("Running 'update_positions_kernel' failed.");
}