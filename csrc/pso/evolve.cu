#include "hip/hip_runtime.h"
#include <cuda/std/limits>

#include "evolve.cuh"
#include "utils.cuh"


#if IS_GLOBAL_BEST_USE_ATOMIC
__global__ void update_best_fits_atomic_kernel(
    double const *x_fits,
    double       *local_best_fits,
    double       *glboal_best_fit,
    size_t       *global_best_idx,
    size_t        num,
    cuda_mutex_t *mutex
){
    __shared__ double glboal_best_fits_smem[BLOCK_DIM_1D];
    __shared__ size_t global_best_idxs_smem[BLOCK_DIM_1D];

    size_t tid = threadIdx.x;
    size_t idx = blockDim.x * blockIdx.x + tid;
    glboal_best_fits_smem[tid] = cuda::std::numeric_limits<double>::max();
    
    for (size_t i = idx; i < num; i += gridDim.x * blockDim.x){
        double x_fit = x_fits[i];
        local_best_fits[i] = min(x_fit, local_best_fits[i]);
        if (x_fit < glboal_best_fits_smem[tid]){
            glboal_best_fits_smem[tid] = x_fit;
            global_best_idxs_smem[tid] = i;
        }
    }
    for (size_t k = blockDim.x >> 1; k > 0; k >>= 1){
        __syncthreads();
        if (tid < k){
            if (glboal_best_fits_smem[tid + k] < glboal_best_fits_smem[tid]){
                glboal_best_fits_smem[tid] = glboal_best_fits_smem[tid + k];
                global_best_idxs_smem[tid] = global_best_idxs_smem[tid + k];
            }
        }
    }
    if (tid == 0){
        lock_kernel_mutex(mutex);
        if (glboal_best_fits_smem[0] < *glboal_best_fit){
            *glboal_best_fit = glboal_best_fits_smem[0];
            *global_best_idx = global_best_idxs_smem[0];
        }
        unlock_kernel_mutex(mutex);
    }
}

#else

__global__ void update_best_fits_reduce_kernel(
    double const *x_fits,
    double       *local_best_fits,
    double       *glboal_best_fits,
    size_t       *global_best_idxs,
    size_t        num
){
    __shared__ double glboal_best_fits_smem[BLOCK_DIM_1D];
    __shared__ size_t global_best_idxs_smem[BLOCK_DIM_1D];

    size_t bid = blockIdx.x;
    size_t tid = threadIdx.x;
    size_t idx = blockDim.x * bid + tid;
    glboal_best_fits_smem[tid] = cuda::std::numeric_limits<double>::max();
    
    for (size_t i = idx; i < num; i += gridDim.x * blockDim.x){
        double x_fit = x_fits[i];
        local_best_fits[i] = min(x_fit, local_best_fits[i]);
        if (x_fit < glboal_best_fits_smem[tid]){
            glboal_best_fits_smem[tid] = x_fit;
            global_best_idxs_smem[tid] = i;
        }
    }
    for (size_t k = blockDim.x >> 1; k > 0; k >>= 1){
        __syncthreads();
        if (tid < k){
            if (glboal_best_fits_smem[tid + k] < glboal_best_fits_smem[tid]){
                glboal_best_fits_smem[tid] = glboal_best_fits_smem[tid + k];
                global_best_idxs_smem[tid] = global_best_idxs_smem[tid + k];
            }
        }
    }
    if (tid == 0){
        glboal_best_fits[bid] = glboal_best_fits_smem[0];
        global_best_idxs[bid] = global_best_idxs_smem[0];
    }
}

__global__ void argmin_global_fits_reduce_kernel(
    double const *glboal_best_fits,
    size_t const *global_best_idxs,
    double       *glboal_best_fit,
    size_t       *global_best_idx,
    size_t        num
){
    __shared__ double glboal_best_fits_smem[BLOCK_DIM_1D];
    __shared__ size_t global_best_idxs_smem[BLOCK_DIM_1D];
    
    size_t tid = threadIdx.x;
    glboal_best_fits_smem[tid] = cuda::std::numeric_limits<double>::max();

    for (size_t i = tid; i < num; i += blockDim.x){
        double part_fit = glboal_best_fits[i];
        size_t part_idx = global_best_idxs[i];
        if (part_fit < glboal_best_fits_smem[tid]){
            glboal_best_fits_smem[tid] = part_fit;
            global_best_idxs_smem[tid] = part_idx;
        }
    }
    for (size_t k = blockDim.x >> 1; k > 0; k >>= 1){
        if (tid < k){
            if (glboal_best_fits_smem[tid + k] < glboal_best_fits_smem[tid]){
                glboal_best_fits_smem[tid] = glboal_best_fits_smem[tid + k];
                global_best_idxs_smem[tid] = global_best_idxs_smem[tid + k];
            }
        }
    }
    if (tid == 0){
        *glboal_best_fit = glboal_best_fits_smem[0];
        *global_best_idx = global_best_idxs_smem[0];
    }
}

#endif

__global__ void assign_local_best_xs(
    double const *xs,
    double const *x_fits,
    double       *local_best_xs,
    double const *local_best_fits,
    size_t        num,
    size_t        dim
){
    size_t nid = blockIdx.x;
    size_t idx = blockDim.x * blockIdx.y + threadIdx.x;

    if (x_fits[nid] > local_best_fits[nid])
        return;

    for (size_t i = idx; i < dim; i += gridDim.y * blockDim.x){
        local_best_xs[nid * dim + i] = xs[nid * dim + i];
    }
}


size_t update_best_fits_cuda(
    double const *x_fits_cuda_ptr,
    double       *local_best_fits_cuda_ptr,
    double       *global_best_fit_cuda_ptr,
    size_t        num
){
    size_t num_block_1d = get_num_block_1d(num);
    size_t global_best_idx;
    size_t *global_best_idx_cuda_ptr;

    hipMalloc(&global_best_idx_cuda_ptr, sizeof(size_t));
    cudaCheckErrors("Failed to allocate memory buffer to 'global_best_idx_cuda_ptr'.");

#if IS_GLOBAL_BEST_USE_ATOMIC
        cuda_mutex_t *mutex;
        cuda_create_mutex(&mutex);
        cudaCheckErrors("Failed to create kernel mutex.");
        update_best_fits_atomic_kernel<<<num_block_1d, BLOCK_DIM_1D>>>(x_fits_cuda_ptr, local_best_fits_cuda_ptr, global_best_fit_cuda_ptr, global_best_idx_cuda_ptr, num, mutex);
        cudaCheckErrors("Failed to run 'update_best_fits_kernel'.");
        cuda_destroy_mutex(mutex);
        cudaCheckErrors("Failed to destroy kernel mutex.");
#else
        if (num_block_1d == 1){
            update_best_fits_reduce_kernel<<<1, BLOCK_DIM_1D>>>(x_fits_cuda_ptr, local_best_fits_cuda_ptr, global_best_fit_cuda_ptr, global_best_idx_cuda_ptr, num);
            cudaCheckErrors("Failed to run 'update_best_fits_reduce_kernel'.");
        }else{
            double *part_global_best_fits_cuda_ptr;
            size_t *part_global_best_idxs_cuda_ptr;
            hipMalloc(&part_global_best_fits_cuda_ptr, num_block_1d * sizeof(double)); cudaCheckErrors("Failed to allocate memory buffer 'part_global_best_fits_cuda_ptr'.");
            hipMalloc(&part_global_best_idxs_cuda_ptr, num_block_1d * sizeof(size_t)); cudaCheckErrors("Failed to allocate memory buffer 'part_global_best_idxs_cuda_ptr'.");
            update_best_fits_reduce_kernel<<<num_block_1d, BLOCK_DIM_1D>>>(x_fits_cuda_ptr, local_best_fits_cuda_ptr, part_global_best_fits_cuda_ptr, part_global_best_idxs_cuda_ptr, num); 
            cudaCheckErrors("Failed to run 'update_best_fits_reduce_kernel'.");
            argmin_global_fits_reduce_kernel<<<1, BLOCK_DIM_1D>>>(part_global_best_fits_cuda_ptr, part_global_best_idxs_cuda_ptr, global_best_fit_cuda_ptr, global_best_idx_cuda_ptr, num); 
            cudaCheckErrors("Failed to run 'argmin_global_fits_reduce_kernel'.");
            hipFree(part_global_best_fits_cuda_ptr); cudaCheckErrors("Failed to free 'part_global_best_fits_cuda_ptr'.");
            hipFree(part_global_best_idxs_cuda_ptr); cudaCheckErrors("Failed to free 'part_global_best_idxs_cuda_ptr'.");
        }
#endif

    hipMemcpy(&global_best_idx, global_best_idx_cuda_ptr, sizeof(size_t), hipMemcpyDeviceToHost);
    cudaCheckErrors("Failed to copy data from 'global_best_idx_cuda_ptr'.");
    hipFree(global_best_idx_cuda_ptr);
    cudaCheckErrors("Failed to free 'global_best_idx_cuda_ptr'.");
    return global_best_idx;
}

size_t update_bests_cuda(
    double const *xs_cuda_ptr,
    double const *x_fits_cuda_ptr,
    double       *local_best_xs_cuda_ptr,
    double       *local_best_fits_cuda_ptr,
    double       *global_best_x_cuda_ptr,
    double       *global_best_fit_cuda_ptr,
    size_t        num,
    size_t        dim
){
    // Update the local & global best fitnesses and find the index of the global best x:
    size_t global_best_idx = update_best_fits_cuda(x_fits_cuda_ptr, local_best_fits_cuda_ptr, global_best_fit_cuda_ptr, num);

    // Assign the local best xs according to the updated local best fitnesses:
    dim3 grid_dims(num, get_num_block_1d(dim));
    dim3 block_dims(BLOCK_DIM_1D);
    assign_local_best_xs<<<grid_dims, block_dims>>>(xs_cuda_ptr, x_fits_cuda_ptr, local_best_xs_cuda_ptr, local_best_fits_cuda_ptr, num, dim);
    cudaCheckErrors("Fail to run 'assign_local_best_xs'.");

    // Assign the global best x according to the index of the global best x:
    hipMemcpy(global_best_x_cuda_ptr, xs_cuda_ptr + global_best_idx * dim, dim * sizeof(double), hipMemcpyDeviceToDevice);
    cudaCheckErrors("Fail to copy global best x.");
    return global_best_idx;
}