#include <iomanip>
#include <sstream>
#include <algorithm>
#include <stdexcept>
#include <thrust/fill.h>
#include <thrust/device_ptr.h>

#include "buffer.cuh"


// start Buffer
Buffer::Buffer(ssize_t nrow, ssize_t ncol, Device device)
    : m_nrow(nrow), m_ncol(ncol), m_device(device)
{
    if (nrow == 0 || ncol == 0){
        m_buffer = nullptr;
        return;
    }
    switch (m_device)
    {
    case Device::CPU:
        m_buffer = new scalar_t[num_elem()];
        break;
    case Device::GPU:
        hipMalloc(&m_buffer, buffer_size()); 
        cudaCheckErrors("Failed to allocate GPU buffer.");
        break;
    }
}
Buffer::Buffer(Buffer const &other)
    : m_nrow(other.m_nrow), m_ncol(other.m_ncol), m_device(other.m_device)
{
    switch (m_device)
    {
    case Device::CPU:
        m_buffer = new scalar_t[num_elem()];
        memcpy(m_buffer, other.m_buffer, buffer_size());
        break;
    case Device::GPU:
        hipMalloc(&m_buffer, buffer_size());
        hipMemcpy(m_buffer, other.m_buffer, buffer_size(), hipMemcpyDeviceToDevice);
        break;
    }
}
Buffer::Buffer(Buffer &&other) noexcept 
    : m_nrow(other.m_nrow), m_ncol(other.m_ncol), m_device(other.m_device), m_buffer(other.m_buffer)
{
    other.m_buffer = nullptr;
    other.m_nrow   = 0;
    other.m_ncol   = 0;
}
Buffer & Buffer::operator=(Buffer const &other){
    if (!is_same_shape(other))
        throw std::runtime_error("Shapes do not match.");
    if (!is_same_device(other))
        throw std::runtime_error("Devices do not match.");
    switch (m_device)
    {
    case Device::CPU:
        memcpy(m_buffer, other.m_buffer, buffer_size());
        break;
    case Device::GPU:
        hipMemcpy(m_buffer, other.m_buffer, buffer_size(), hipMemcpyDeviceToDevice);
        break;
    }
    return *this;
}
Buffer & Buffer::operator=(Buffer &&other) noexcept {
    _release();
    m_buffer       = other.m_buffer;
    m_nrow         = other.m_nrow;
    m_ncol         = other.m_ncol;
    m_device       = other.m_device;
    other.m_buffer = nullptr;
    other.m_nrow   = 0;
    other.m_ncol   = 0;
    return *this;
}
Buffer::~Buffer(){
    _release();
}

void Buffer::set_value(ssize_t row, ssize_t col, scalar_t val) {
    switch (m_device)
    {
    case Device::CPU:
        m_buffer[index_at(row, col)] = val;
        break;
    case Device::GPU:
        hipMemcpy(m_buffer + index_at(row, col), &val, sizeof(scalar_t), hipMemcpyHostToDevice);
        break;
    }
}
scalar_t Buffer::get_value(ssize_t row, ssize_t col) const {
    scalar_t res;
    switch (m_device)
    {
    case Device::CPU:
        res = m_buffer[index_at(row, col)];
        break;
    case Device::GPU:
        hipMemcpy(&res, m_buffer + index_at(row, col), sizeof(scalar_t), hipMemcpyDeviceToHost);
        break;
    }
    return res;
}
scalar_t Buffer::operator()(ssize_t row, ssize_t col) const {
    return get_value(row, col);
}

scalar_t * Buffer::data_ptr() const {
    return m_buffer;
}
scalar_t const * Buffer::cdata_ptr() const {
    return m_buffer;
}

Device Buffer::device() const {
    return m_device;
}
shape_t Buffer::shape() const {
    return {m_nrow, m_ncol};
}
ssize_t Buffer::nrow() const {
    return m_nrow;
}
ssize_t Buffer::ncol() const {
    return m_ncol;
}
ssize_t Buffer::num_elem() const {
    return m_nrow * m_ncol;
}
ssize_t Buffer::buffer_size() const {
    return m_nrow * m_ncol * sizeof(scalar_t);
}
ssize_t Buffer::index_at(ssize_t row, ssize_t col) const {
    return row * m_ncol + col;
}
bool Buffer::is_same_shape(Buffer const &other) const {
    return m_nrow == other.nrow() && m_ncol == other.ncol();
}
bool Buffer::is_same_device(Buffer const &other) const {
    return m_device == other.device();
}
std::string Buffer::to_string() const {
    std::stringstream ss;
    ss << "<Buffer shape=(" << m_nrow << ", " << m_ncol << ") device=" << ((m_device == Device::CPU)? "CPU": "GPU") << " @" << (uintptr_t)this << ">";
    return ss.str();
}

void Buffer::to(Device device){
    if (m_device == device)
        return;
    scalar_t *new_buffer;
    switch (m_device)
    {
    case Device::CPU:
        hipMalloc(&new_buffer, buffer_size());
        hipMemcpy(new_buffer, m_buffer, buffer_size(), hipMemcpyHostToDevice);
        delete[] m_buffer;
        break;
    case Device::GPU:
        new_buffer = new scalar_t[num_elem()];
        hipMemcpy(new_buffer, m_buffer, buffer_size(), hipMemcpyDeviceToHost);
        hipFree(m_buffer);
        break;
    }
    m_buffer = new_buffer;
    m_device = device;
}
void Buffer::fill(scalar_t val){
    switch (m_device)
    {
    case Device::CPU:
        std::fill_n(m_buffer, num_elem(), val);
        break;
    case Device::GPU:
        thrust::device_ptr<scalar_t> dev_ptr(m_buffer);
        thrust::fill_n(dev_ptr, num_elem(), val); 
        break;
    }
}
void Buffer::show(){
    for (ssize_t i = 0; i < m_nrow; i++)
    for (ssize_t j = 0; j < m_ncol; j++){
        if (j == m_ncol - 1)
            std::cout << std::fixed << std::setprecision(8) << get_value(i, j) << std::endl;
        else
            std::cout << std::fixed << std::setprecision(8) << get_value(i, j) << ", ";
    }
}
void Buffer::clear(){
    _release();
    m_buffer = nullptr;
    m_nrow   = 0;
    m_ncol   = 0;
}

void Buffer::_release(){
    if (!m_buffer)
        return;
    switch (m_device)
    {
    case Device::CPU:
        delete[] m_buffer;
        break;
    case Device::GPU:
        hipFree(m_buffer); 
        cudaCheckErrors("Failed to free GPU buffer.");
        break;
    }
}

void Buffer::copy_to_numpy(ndarray_t<scalar_t> &out) const {
    ssize_t buf_buffer_size = buffer_size();
    ssize_t npy_buffer_size = out.nbytes();

    if (npy_buffer_size != buf_buffer_size)
        throw std::runtime_error("Size of numpy array does not match this buffer.");
    
    switch (m_device)
    {
    case Device::CPU:
        memcpy(out.mutable_data(), m_buffer, buf_buffer_size);
        break;
    case Device::GPU:
        hipMemcpy(out.mutable_data(), m_buffer, buf_buffer_size, hipMemcpyDeviceToHost);
        break;
    }
}
void Buffer::copy_from_numpy(ndarray_t<scalar_t> const &src) const {
    ssize_t buf_buffer_size = buffer_size();
    ssize_t npy_buffer_size = src.nbytes();

    if (npy_buffer_size != buf_buffer_size)
        throw std::runtime_error("Size of numpy array does not match this buffer.");
    
    switch (m_device)
    {
    case Device::CPU:
        memcpy(m_buffer, src.data(), buf_buffer_size);
        break;
    case Device::GPU:
        hipMemcpy(m_buffer, src.data(), buf_buffer_size, hipMemcpyHostToDevice);
        break;
    }
}
// end Buffer

// start CURANDStates
CURANDStates::CURANDStates(ssize_t size, unsigned long long seed) 
    : m_size(size) 
{
    curand_setup(size, seed, &m_buffer);
}
CURANDStates::CURANDStates(CURANDStates const &other)
    : m_size(other.m_size)
{
    hipMalloc(&m_buffer, buffer_size());
    hipMemcpy(m_buffer, other.m_buffer, buffer_size(), hipMemcpyDeviceToDevice);
}
CURANDStates::CURANDStates(CURANDStates &&other) noexcept
    : m_size(other.m_size), m_buffer(other.m_buffer)
{
    other.m_buffer = nullptr;
    other.m_size   = 0;
}
CURANDStates & CURANDStates::operator=(CURANDStates const &other){
    if (m_size != other.m_size)
        throw std::runtime_error("Shapes do not match.");
    hipMemcpy(m_buffer, other.m_buffer, buffer_size(), hipMemcpyDeviceToDevice);
    m_size = other.m_size;
    return *this;
}
CURANDStates & CURANDStates::operator=(CURANDStates &&other) noexcept {
    _release();
    m_buffer       = other.m_buffer;
    m_size         = other.m_size;
    other.m_buffer = nullptr;
    other.m_size   = 0;
    return *this;
}
CURANDStates::~CURANDStates(){
    _release();
}

cuda_rng_t * CURANDStates::data_ptr() const {
    return m_buffer;
}
cuda_rng_t const * CURANDStates::cdata_ptr() const {
    return m_buffer;
}

ssize_t CURANDStates::num_elem() const {
    return m_size;
}
ssize_t CURANDStates::buffer_size() const {
    return m_size * sizeof(cuda_rng_t);
}
std::string CURANDStates::to_string() const {
    std::stringstream ss;
    ss << "<CURANDStates size=" << m_size << " device=GPU @" << (uintptr_t)this << ">";
    return ss.str();
}

void CURANDStates::clear(){
    _release();
    m_buffer = nullptr;
    m_size   = 0;
}

void CURANDStates::_release(){
    if (!m_buffer)
        return;
    curand_destroy(m_buffer);
}
// end CURANDStates