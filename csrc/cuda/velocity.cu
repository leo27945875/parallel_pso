#include "hip/hip_runtime.h"
#include "velocity.cuh"


__global__ void update_velocities_kernel(
    double       *vs, 
    double const *xs, 
    double const *local_best_xs, 
    double const *global_best_x,
    double        w,
    double        c0,
    double        c1,
    size_t        num, 
    size_t        dim,
    hiprandState   *rng_states
){
    size_t nid = blockIdx.x;
    size_t bid = blockIdx.y;
    size_t tid = threadIdx.x;
    size_t idx = bid * blockDim.x + tid;

    hiprandState thread_rng_state = rng_states[nid * dim + idx];

    for (size_t i = idx; i < dim; i += gridDim.y * blockDim.x){
        // Load data into local memory:
        double v = vs[nid * dim + i];
        double x = xs[nid * dim + i];
        double lbest_x = local_best_xs[nid * dim + i];
        double gbest_x = global_best_x[i];

        // Calculate new velocities:
        vs[nid * dim + i] = (
            w * v + 
            c0 * hiprand_uniform_double(&thread_rng_state) * (lbest_x - x) + 
            c1 * hiprand_uniform_double(&thread_rng_state) * (gbest_x - x)
        );
    }
    // Return new rng state:
    rng_states[nid * dim + idx] = thread_rng_state;
}

__global__ void update_velocities_with_sum_pow2_kernel(
    double       *vs, 
    double const *xs, 
    double const *local_best_xs, 
    double const *global_best_x,
    double       *sum_pow2_res,
    double        w,
    double        c0,
    double        c1,
    size_t        num, 
    size_t        dim,
    hiprandState   *rng_states
){
    __shared__ double p_smem[BLOCK_DIM_1D];

    size_t nid = blockIdx.x;
    size_t bid = blockIdx.y;
    size_t tid = threadIdx.x;
    size_t idx = bid * blockDim.x + tid;

    hiprandState thread_rng_state = rng_states[nid * dim + idx];

    p_smem[tid] = 0.;
    for (size_t i = idx; i < dim; i += gridDim.y * blockDim.x){
        // Load data into local memory:
        double v = vs[nid * dim + i];
        double x = xs[nid * dim + i];
        double lbest_x = local_best_xs[nid * dim + i];
        double gbest_x = global_best_x[i];

        // Load data into shared memory:
        p_smem[tid] += v * v;

        // Calculate new velocities:
        vs[nid * dim + i] = (
            w * v + 
            c0 * hiprand_uniform_double(&thread_rng_state) * (lbest_x - x) + 
            c1 * hiprand_uniform_double(&thread_rng_state) * (gbest_x - x)
        );
    }
    // Sum the squares:
    for (size_t k = blockDim.x / 2; k > 0; k >>= 1){
        __syncthreads();
        if (tid < k)
            p_smem[tid] += p_smem[tid + k];
    }
    if (tid == 0)
        atomicAdd(sum_pow2_res + nid, p_smem[0]);
    
    // Return new rng state:
    rng_states[nid * dim + idx] = thread_rng_state;
}

__global__ void norm_clip_velocities_kernel(
    double *vs, 
    double *sum_pow2_res,
    double  v_max,
    size_t  num, 
    size_t  dim
){
    size_t nid = blockIdx.x;
    size_t bid = blockIdx.y;
    size_t tid = threadIdx.x;
    size_t idx = bid * blockDim.x + tid;

    double norm = sqrt(sum_pow2_res[nid]); // 'Broadcast' mechanism (see https://forums.developer.nvidia.com/t/accessing-same-global-memory-address-within-warps/66574)
    if (norm <= v_max)
        return;
    for (size_t i = idx; i < dim; i += gridDim.y * blockDim.x){
        vs[nid * dim + i] /= norm;
    }
}

void update_velocities_cuda(
    double       *vs_cuda_ptr, 
    double const *xs_cuda_ptr, 
    double const *local_best_xs_cuda_ptr, 
    double const *global_best_x_cuda_ptr,
    double       *sum_pow2_cuda_ptr,
    double        w,
    double        c0,
    double        c1,
    double        v_max,
    size_t        num, 
    size_t        dim,
    hiprandState   *rng_states
){
    size_t num_block_per_x = get_num_block_per_x(dim);
    dim3 grid_dims(num, num_block_per_x);
    dim3 block_dims(BLOCK_DIM_1D);
    if (v_max <= 0.){
        update_velocities_kernel<<<grid_dims, block_dims>>>(
            vs_cuda_ptr, xs_cuda_ptr, local_best_xs_cuda_ptr, global_best_x_cuda_ptr, w, c0, c1, num, dim, rng_states
        );
        cudaCheckErrors("Running 'update_velocities_kernel' failed.");
    }else{
        update_velocities_with_sum_pow2_kernel<<<grid_dims, block_dims>>>(
            vs_cuda_ptr, xs_cuda_ptr, local_best_xs_cuda_ptr, global_best_x_cuda_ptr, sum_pow2_cuda_ptr, w, c0, c1, num, dim, rng_states
        );
        cudaCheckErrors("Running 'update_velocities_with_sum_pow2_kernel' failed.");
        norm_clip_velocities_kernel<<<grid_dims, block_dims>>>(
            vs_cuda_ptr, sum_pow2_cuda_ptr, v_max, num, dim
        );
        cudaCheckErrors("Running 'norm_clip_velocities_kernel' failed.");
    }
}