#include "hip/hip_runtime.h"
#include "position.cuh"
#include "utils.cuh"


__global__ void update_positions_kernel(
    double       *xs,
    double const *vs,
    double        x_min,
    double        x_max,
    size_t        num,
    size_t        dim
){
    size_t nid = blockIdx.x;
    size_t idx = blockIdx.y * blockDim.x + threadIdx.x;
    for (size_t i = idx; i < dim; i += gridDim.y * blockDim.x)
        xs[nid * dim + i] = min(
            x_max, 
            max(
                x_min, 
                xs[nid * dim + i] + vs[nid * dim + i]
            )
        );
}

void update_positions_cuda(
    double       *xs,
    double const *vs,
    double        x_min,
    double        x_max,
    size_t        num,
    size_t        dim
){
    size_t num_block_per_x = get_num_block_per_x(dim);
    dim3 grid_dims(num, num_block_per_x);
    dim3 block_dims(BLOCK_DIM_1D);
    update_positions_kernel<<<grid_dims, block_dims>>>(
        xs, vs, x_min, x_max, num, dim
    );
    cudaCheckErrors("Running 'update_positions_kernel' failed.");
}