#include "hip/hip_runtime.h"
#include <cmath>
#include "levy.cuh"
#include "utils.cuh"


__host__ __device__ double levy_w_func(double x){
    return 1. + 0.25 * (x - 1.);
}
__host__ __device__ double levy_head_func(double x){
    return pow2(sin(M_PI * levy_w_func(x)));
}
__host__ __device__ double levy_tail_func(double x){
    double w = levy_w_func(x);
    return pow2(w - 1.) * (1. + pow2(sin(2. * M_PI * w)));
}
__host__ __device__ double levy_middle_func(double x){
    double w = levy_w_func(x);
    return pow2(w - 1.) * (1. + 10. * pow2(sin(M_PI * w + 1.)));
}

__global__ void levy_function_kernel(double const *xs, double *out, size_t num, size_t dim){
    __shared__ double smem[BLOCK_DIM_1D];
    size_t nid = blockIdx.x;
    size_t bid = blockIdx.y;
    size_t tid = threadIdx.x;
    size_t idx = bid * blockDim.x + tid;

    if (nid >= num)
        return;

    smem[tid] = 0.;
    for (size_t i = idx; i < dim; i += gridDim.y * blockDim.x){
        double x = xs[nid * dim + i];
        if (i == 0)
            smem[tid] += levy_head_func(x) + levy_middle_func(x);
        else if (i == dim - 1)
            smem[tid] += levy_tail_func(x);
        else
            smem[tid] += levy_middle_func(x);
    }
    
    for (size_t k = blockDim.x / 2; k > 0; k >>= 1){
        __syncthreads();
        if (tid < k)
            smem[tid] += smem[tid + k];
    }
    if (tid == 0)
        atomicAdd(out + nid, smem[0]);
}

void levy_function_cuda(double const *xs_cuda_ptr, double *out_cuda_ptr, size_t num, size_t dim){
    size_t num_block_per_x = get_num_block_1d(dim);
    dim3 grid_dims(num, num_block_per_x);
    dim3 block_dims(BLOCK_DIM_1D);
    levy_function_kernel<<<grid_dims, block_dims>>>(xs_cuda_ptr, out_cuda_ptr, num, dim); 
    cudaCheckErrors("Running 'levy_function_kernel' failed.");
}

void levy_function_cpu(double const *xs, double *out, size_t num, size_t dim){
    for (size_t nid = 0; nid < num; nid++){
        out[nid] = 0.;
        for (size_t idx = 0; idx < dim; idx++){
            double x = xs[nid * dim + idx];
            if (idx == 0)
                out[nid] += levy_head_func(x) + levy_middle_func(x);
            else if (idx == dim - 1)
                out[nid] += levy_tail_func(x);
            else
                out[nid] += levy_middle_func(x);
        }
    }
}