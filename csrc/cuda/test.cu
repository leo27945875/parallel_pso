#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <cstdlib>

#include "levy.cuh"
#include "utils.cuh"


void test_levy_function(
    size_t num         = 1000,
    size_t dim         = 500,
    double tol         = 1e-5,
    bool   is_show_out = true
){

    double *xs, *out_cpu, *out_cuda;
    double *d_xs, *d_out;

    srand(time(NULL));

    xs       = new double[num * dim];
    out_cpu  = new double[num];
    out_cuda = new double[num];

    for (size_t i = 0; i < num * dim; i++) 
        xs[i] = rand_number();
    
    levy_function_cpu(xs, out_cpu, num, dim);

    hipMalloc(&d_xs, num * dim * sizeof(double));
    hipMalloc(&d_out, num * sizeof(double));
    hipMemcpy(d_xs, xs, num * dim * sizeof(double), hipMemcpyHostToDevice);
    levy_function_cuda(d_xs, d_out, num, dim);
    hipMemcpy(out_cuda, d_out, num * sizeof(double), hipMemcpyDeviceToHost);

    if (is_show_out){
        std::cout << "\nLevy results: (CPU)" << std::endl;
        for (size_t i = 0; i < num; i++)
            std::cout << out_cpu[i] << ", ";
        std::cout << std::endl;

        std::cout << "\nLevy results: (CUDA)" << std::endl;
        for (size_t i = 0; i < num; i++)
            std::cout << out_cuda[i] << ", ";
        std::cout << std::endl;
    }

    bool is_close = true;
    for (size_t i = 0; i < num; i++){
        is_close = is_close && abs(out_cpu[i] - out_cuda[i]) < tol;
    }
    std::cout << "\nis_close = " << is_close << std::endl;

    hipFree(d_xs);
    hipFree(d_out);
    delete[] xs;
    delete[] out_cpu;
    delete[] out_cuda;
}


int main(){

    test_levy_function();
    
    return 0;
}